
#include <hip/hip_runtime.h>
extern "C"
__global__ void compute(
    unsigned int width,
    unsigned int height,
    unsigned char* current,
    unsigned char* next)
{

    int column = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (column < width && row < height)
    {
        int cells = 0;
        for (int nRow = row - 1; nRow <= row + 1; nRow++) {
            if (0 <= nRow && nRow < height) {

                for (int nColumn = column - 1; nColumn <= column + 1; nColumn++) {
                    if (0 <= nColumn && nColumn < width) {

                        cells += current[nRow * width + nColumn];

                    }
                }
            }
        }

        //recycling variable
        column = (row * width) + column;

        switch (cells)
        {
        case 3:
            next[column] = 1;
            break;
        case 4:
            next[column] = current[column];
            break;
        default:
            next[column] = 0;
        }
    }
}